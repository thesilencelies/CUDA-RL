#include "hip/hip_runtime.h"
#include "rlagent.cuh"
#include <vector>
#include <sstream>

namespace mancalaCuda
{
    //cuda functions

/*
every cuda block has it's own simulation of the game it's playing through
each sim is just an array npits*2 + 2 large of ints plus a flag indicating who's turn it is
*/
	__device__ bool take_turn(board_state & bs, int action, bool & turnval)
	{
        //parameters for the turn
        int start_index = turnval*(nPits_player +1) + action;
        int pool_index = (turnval + 1) *(nPits_player + 1) - 1;
        //take the turn
        int beads = bs.pits[start_index];
        bs.pits[start_index] = 0;
        int index = start_index;
        for (int j =0; j < beads; j++)
        {
            index ++;
            if (index >= nPits_total)
            {
                index = 0;
            }
            bs.pits[index] ++;
        }
        if (index != pool_index)
        {
            turnval = !turnval;
        }
        //empty pot handling
        if (bs.pits[index] == 1 && index >= turnval*(nPits_player + 1) && index < pool_index)
        {
            int opp_index = (nPits_player * 2 - index) % nPits_total;
            bs.pits[pool_index] += bs.pits[opp_index] + bs.pits[index];
            bs.pits[opp_index] = 0;
            bs.pits[index] = 0;
        }

        for(int j = 0; j < nPits_player; j++)
        {
            if (bs.player1pits[j] > 0 || bs.player2pits[j] > 0)
            {
                return false;
            }
        }
        return true;
	}

    __device__ int chooseAction(board_state bs, bool player, const float* QMat)
    {
        int stateIndex = 0;
        auto ownPits = player? bs.player2pits : bs.player1pits;
        for(int i = 0 ; i < nPits_player; i++)
        {
            //0 state is the empty pit indicator
            int pitIndex = (ownPits[i] == 0 ? 0 : (i + ownPits[i] % nPits_total) + 1);
            stateIndex = stateIndex * nStates_pit + pitIndex;
        }
        auto oppPits = player? bs.player1pits : bs.player2pits;
        for(int i = 0 ; i < nPits_player; i++)
        {
            int pitIndex = (oppPits[i] == 0 ? 0 : (i + oppPits[i] % nPits_total) + 1);
            stateIndex = stateIndex * nStates_pit + pitIndex;
        }
        stateIndex = stateIndex * nPits_player;
        //explortation
        //start deterministic
        //choose based on Qmat
        float maxQ = -100000;
        int rval = 0;
        //for test just return first valid
        int playerInd = player ? nPits_player + 1 : 0;
        for(int i = 0; i < nPits_player; i++)
        {
            float qval = QMat[stateIndex + i];
            if(bs.pits[playerInd + i] > 0 &&  qval > maxQ)
            {
                rval = i;
                maxQ = qval;
            }
        }
        return rval;
    }

    __global__ void playGame(int num_sims, int nturns, turn_record * results, const float * QMat)
    {
		int run_index = blockIdx.x * blockDim.x + threadIdx.x;
		int run_stride = blockDim.x * gridDim.x;
		for (int i = run_index; i < num_sims; i += run_stride)
		{
            bool player = false;
            bool newgame = true;
            board_state board;
            for(int t = 0; t < nturns; t++)
            {
                if(newgame)
                {
                    for(int p = 0; p < nPits_player; p++)
                    {
                        board.player1pits[p] = nSeeds;
                        board.player2pits[p] = nSeeds;
                    }
                    board.player1pool = 0;
                    board.player2pool = 0;
                    newgame = false;
                }

                results[nturns*i + t].state = board;
                results[nturns*i + t].player = player;
                 //if a game finishes start a new one, we can finish the sim mid step
                int action = chooseAction(board, player, QMat);
                newgame = take_turn(board, action, player);
                results[nturns*i + t].action = action;
                if(newgame)
                {
                    for(int p = 0; p < nPits_player; p++)
                    {
                        board.player1pool += board.player1pits[p];
                        board.player2pool += board.player2pits[p];
                    }
                    results[nturns * i + t].reward = board.player1pool > board.player2pool ? 1 : 
                                                (board.player1pool < board.player2pool ? -1 : -2);
                }
                else
                {
                   results[nturns*i + t].reward = 0;
                }
            }
        }
    }

    //class functions

    void RLagent::parseBoardState(board_state& state, std::ostream & stream)
    {
        stream << "    |";
        for(int i = 0; i < nPits_player; i++)
        {
            stream << state.player1pits[i]  << "|"; 
        }
        stream << std::endl;
        stream << " |" << state.player1pool << "| ";
        for(int i = 0; i < nPits_player; i++)
        {
            stream << "  ";
        }
        stream << "|" << state.player2pool << "| ";
        stream << std::endl;
        stream << "    |";
        for(int i = 0; i < nPits_player; i++)
        {
            stream << state.player2pits[i]  << "|"; 
        }
        stream << std::endl;
    }

    RLagent::RLagent(int num_sims = 10000, int num_turns = 200)
	{
		name = "rlagent";
        this->num_sims = num_sims;
        this->num_turns = num_turns;
        num_records = num_sims*num_turns;
        record_size = num_records * sizeof(turn_record);
        num_states = nPits_player* pow(nPits_total +1, nPits_player*2);
        state_size = num_states * sizeof(float);

        h_turnRecord.resize(num_records);
        h_Qvals.resize(num_states);
        hipMalloc(&d_Qvals, state_size);
        hipMalloc(&d_turnRecord, record_size);
        hipMemcpy(d_Qvals, h_Qvals.data(), state_size, hipMemcpyHostToDevice);
	}

    RLagent::~RLagent()
    {
        hipFree(d_turnRecord);
        hipFree(d_Qvals);
    }
	
	std::string RLagent::GetName()
	{
		return name;
	}



    void RLagent::RunStep()
    {
       int threadsPerBlock = 32;
       int blocksPerGrid = (num_sims + threadsPerBlock - 1) / threadsPerBlock;

        playGame<<<blocksPerGrid, threadsPerBlock>>>(num_sims, num_turns, d_turnRecord, d_Qvals );
    }

    void RLagent::TrainStep()
    {        
        hipMemcpy(h_Qvals.data(), d_Qvals, state_size, hipMemcpyDeviceToHost);
    }

    std::string RLagent::PrintRun()
    {
        hipMemcpy(h_turnRecord.data(), d_turnRecord, record_size, hipMemcpyDeviceToHost);
        std::stringstream outStream;
        for(int i = 0; i < num_turns; i++)
        {
            outStream << "turn " << i << " action: " << h_turnRecord[i].action << " player: " <<
                h_turnRecord[i].player << " reward: " << h_turnRecord[i].reward << std::endl;

            parseBoardState(h_turnRecord[i].state, outStream); 
            outStream << std::endl;
        }
        return outStream.str();
    }
}